#include "hip/hip_runtime.h"
//          Copyright Oliver Kowalke 2017.
// Distributed under the Boost Software License, Version 1.0.
//    (See accompanying file LICENSE_1_0.txt or copy at
//          http://www.boost.org/LICENSE_1_0.txt)

#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>

#include "animation.h"

constexpr unsigned int DIM = 2048;
constexpr unsigned int STATES = 15;

__global__
void cell( unsigned int * in, unsigned int * out) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ( x < DIM && y < DIM) {
        int offset = x + y * DIM;
        unsigned int t = in[x+(y+1)%DIM*DIM];
        unsigned int l = in[(0 == x) ? DIM-1+y*DIM : (x-1)%DIM+y*DIM];
        unsigned int c = in[offset];
        unsigned int r = in[(x+1)%DIM+y*DIM];
        unsigned int b = in[(0 == y) ? x+(DIM-1)*DIM : x+(y-1)%DIM*DIM];
        unsigned int tm = t % STATES;
        unsigned int lm = l % STATES;
        unsigned int cm = (c + 1) % STATES;
        unsigned int rm = r % STATES;
        unsigned int bm = b % STATES;
        if ( tm == cm) { 
            out[offset] = t;
        } else if ( lm == cm) {
            out[offset] = l;
        } else if ( rm == cm) {
            out[offset] = r;
        } else if ( bm == cm) {
            out[offset] = b;
        }
    }
}

__device__
std::tuple< unsigned char, unsigned char, unsigned char >
rgb_from_int( unsigned int state) {
    float f = static_cast< float >(state)/(STATES-1);
    return std::make_tuple(
            static_cast< unsigned char >(255*f),
            static_cast< unsigned char >(0),
            static_cast< unsigned char >(255*(1-f)));
}

__global__
void map( unsigned char* optr, unsigned int * inSrc, unsigned int const* outSrc) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ( x < DIM && y < DIM) {
        int offset = x + y * DIM;
        int state = outSrc[offset];
        inSrc[offset] = state;
        auto t = rgb_from_int( state);
        optr[4 * offset + 0] = std::get<0>( t);
        optr[4 * offset + 1] = std::get<1>( t);
        optr[4 * offset + 2] = std::get<2>( t);
        optr[4 * offset + 3] = 255;
    }
}

int main() {
    constexpr std::size_t size = DIM * DIM * sizeof( unsigned int);
    unsigned int * inSrc = nullptr;
    unsigned int * outSrc = nullptr;
    hipMallocManaged( & inSrc, size);
    hipMallocManaged( & outSrc, size);
    std::minstd_rand generator;
    std::uniform_int_distribution<> distribution{ 0, STATES-1 };
    for ( unsigned int i = 0; i < DIM*DIM; ++i) {
        inSrc[i] = distribution( generator); 
        outSrc[i] = inSrc[i];
    }
    animation image{ DIM, DIM };
    unsigned int x = std::ceil( DIM/32.0);
    const dim3 dim_grid{ x, x };
    const dim3 dim_block{ 32, 32 };
    image.display_and_exit(
        [dim_grid,dim_block,&image,size,inSrc,outSrc](unsigned int) mutable {
            cell<<< dim_grid, dim_block >>>( inSrc, outSrc);
            map<<< dim_grid, dim_block >>>( image.get_ptr(), inSrc, outSrc);
            hipDeviceSynchronize();
        });
    hipFree( inSrc);
    hipFree( outSrc);
    return EXIT_SUCCESS;
}
